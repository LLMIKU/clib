#include "math_functions.h"
#include "common.h"
#include <cmath>
#include <cstdlib>
#include <cstring>

template <>
void c_gpu_gemm<float>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  (hipblasSgemm(C_singleton::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void c_gpu_gemm<double>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  (hipblasDgemm(C_singleton::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void c_gpu_gemv<float>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const float alpha, const float* A, const float* x,
    const float beta, float* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  (hipblasSgemv(C_singleton::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void c_gpu_gemv<double>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const double alpha, const double* A, const double* x,
    const double beta, double* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  (hipblasDgemv(C_singleton::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void c_gpu_axpy<float>(const int N, const float alpha, const float* X,
    float* Y) {
  (hipblasSaxpy(C_singleton::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

template <>
void c_gpu_axpy<double>(const int N, const double alpha, const double* X,
    double* Y) {
  (hipblasDaxpy(C_singleton::cublas_handle(), N, &alpha, X, 1, Y, 1));
}
